#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
/*
 * Author: MARTIN VIVANCO
 * ID: A01701167
*/
#define N (4096*4096)
#define HILOS_POR_BLOQUE 512

__global__ void matrixmul(int *a, int *b, int *c, int awith, int aheight, int bwith, int bheight){
	int index = threadIdx.x + blockIdx.x*blockDim.x;// get index of the thread
  int n = (bwith+1)*(aheight+1); //get length of result
	if (index < n){ // if thread id is smaller than c size
		int col = index%(bwith+1);//get col of a matrix
    int row = index/(bwith+1);//get row of b matrix
    int total = 0; // mult and sum of row per col
    for (int i = 0; i < (awith+1); i++) {
      total += a[i+row*(awith+1)]*b[col+i*(bwith+1)];
    }
    c[index] = total;// save total in the result matrix
	}
}

void examplem1(int *a){
  int j = 1;
	for (int i = 0; i < 6; i++){
		a[i] = j;
    j++;
	}
}
void examplem2(int *a){
	int j = 6;
	for (int i = 0; i < 4; i++){
		a[i] = j;
    j++;
	}
  a[4] = 0;
  a[5] = 1;
}
int main(){
	int *a, *b, *c;// hosts matrix
	int *d_a, *d_b, *d_c;// device matrix
  int awith, aheight, bwith, bheight; // with and heigth of matrix a and b
  printf("please give me a with and heigth starting 0\n");//For predefined example 1 2 or 2 1
	scanf("%i %i", &awith, &aheight);
  printf("please give me b with and heigth starting 0\n");//For predefined example 2 1 or 1 2
	scanf("%i %i", &bwith, &bheight);
	int tama = (awith+1)*(aheight+1)*sizeof(int);// sizes not starting in 0
  int tamb = (bwith+1)*(bheight+1)*sizeof(int);
  int tamc = (bwith+1)*(aheight+1)*sizeof(int);// size of the result
	//reserva memoria en DEVICE gpu
	hipMalloc((void**)&d_a, tama);
	hipMalloc((void**)&d_b, tamb);
	hipMalloc((void**)&d_c, tamc);
	// reserva de memoria en HOST cpu
	a = (int*)malloc(tama);
	b = (int*)malloc(tamb);
	c = (int*)malloc(tamc);
	//init with predefined examples
	examplem1(a);
	examplem2(b);
	//copy values from cpu to gpu
	hipMemcpy(d_a, a, tama, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, tamb, hipMemcpyHostToDevice);
	//ejecute kernel
	matrixmul << <N / HILOS_POR_BLOQUE, HILOS_POR_BLOQUE >> >(d_a, d_b, d_c, awith, aheight, bwith, bheight);
	//copy result from gpu to cpu
	hipMemcpy(c, d_c, tamc, hipMemcpyDeviceToHost);
	// print result
	int cont = 0;
  for (int i = 0; i < (bwith+1)*(aheight+1); i++) {
    printf(" %i ",c[i]);
    if (cont == bwith) {
      printf("\n");
      cont = -1;
    }
    cont++;
  }

	//free memory
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
